
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cmath>
#include <cstdlib>

using namespace std;




int * make_array_2_to_n(int n) {

	int * array = (int *) malloc((n-1) * sizeof(int));
	for (int i = 0; i < (n-1); i++) {
		array[i] = 1;
	}
	return array;
}

void print_array(int * arr, int n) {

	for (int i = 0; i < (n-1); i++) {
		cout << (i+2) << " " << arr[i] << endl;
	}

}

void print_prime(int * arr, int n) {

	for (int i = 0; i < (n-1); i++) {
		if (arr[i] == 1) {
			cout << (i+2) << endl;
		}
	}
}

void diff_prime(int * arr1, int * arr2, int n) {

	int flag = 1;
	for (int i = 0; i < (n-1); i++) {
		if (arr1[i] != arr2[i]) {
			if (flag == 1) { flag = 0; }
			cout << "Arrays are different\n";
			cout << (i+2) << " " << arr1[i] << " " << arr2[i] << endl;
		}
	}
	if (flag == 1) {
		cout << "Arrays are the same\n";
	}
}	


void seq_sieve(int * arr, int n) {

	int sqrt_n = int(ceil(sqrt(int(n))));
	int i_sqr;	

	for (int i = 2; i <= sqrt_n; i++) {
		if (arr[i-2] == 1) {
			i_sqr = i * i;
			for (int j = i_sqr; j <= n; j+=i) {
				arr[j - 2] = 0;
			}
		}
	}
}

__global__
void par_sieve(int * d_arr, int n, int sqrt_n) {

	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	__syncthreads();

	for (int i = 2; i <= sqrt_n; i++) {
		if (tid <= sqrt_n) {	
			if (d_arr[i-2] == 1) {
				for (int j = 0; j < n; j+=sqrt_n) {
					if ((j + tid + (2*i) - 2 < n) && (((j + tid + i) % i) == 0)) {
						d_arr[j + tid + (2*i) - 2] = 0;
					}
				}
			}
		}
	}

}



int main(int argc, char** argv) {

	if (argc != 2) {
		cout << "Takes one argument - n, positive integer - to calculate the number of primes at most n\n";
	}

	int n = atoi(argv[1]);

	int * seq_array = make_array_2_to_n(n);
	//print_array(seq_array, n);
	seq_sieve(seq_array, n);
	//print_prime(seq_array, n);

	int sqrt_n = int(ceil(sqrt(int(n))));
	int * par_array = make_array_2_to_n(n);
	int * d_par_array;

	hipMalloc((void**)&d_par_array, sizeof(int) * (n-1));
	hipMemcpy((void*)d_par_array, (void*)par_array, sizeof(int) * (n-1), hipMemcpyHostToDevice);

	int tpb = 1024;
	int nblocks = n / tpb + 1;
	
	cout << "parallel \n\n\n";

	par_sieve<<<nblocks, tpb>>>(d_par_array, n, sqrt_n);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}
	

	hipMemcpy((void*)par_array, (void*)d_par_array, sizeof(int) * (n-1), hipMemcpyDeviceToHost);
	//print_prime(par_array, n);

	//diff_prime(seq_array, par_array, n);

	return 0;

}
