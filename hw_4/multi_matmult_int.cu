
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_DIM 20
#define SCALING_FACTOR 256
#define NUM_THREADS 1024
#define MOD_BASE 10007

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int find_max(int * arr, int num_elem)
{
    int max = 0;
    for (int i = 0; i < num_elem; i++) {
        if (arr[i] > max) {
            max = arr[i];
        }
    }
    return max;
}

int * def_mat_dim(int k)
{
    int * dim = (int *) malloc(k * sizeof(int));
    int i;
    srand(time(NULL));

    for (i = 0; i < k; i++)
    {
        dim[i] = (rand() % MAX_DIM) + 1;
        //printf("%d\n", dim[i]);
    }
    return dim;
}

int * equipartition(int k, int blocks) 
{
    float div = float(k) / float(blocks);
    int div_int = int(div);
    float rem = div - float(div_int);

    int * partition = (int *) malloc((blocks + 1) * sizeof(int));
    srand(time(NULL));

    partition[0] = 0;
    partition[blocks] = k-1;

    int cur_index = 0;
    float round_factor = 0.0;
    for (int i = 1; i < blocks; i++) {
        cur_index += div;
        round_factor = float(rand()) / float(RAND_MAX);
        if (round_factor < rem) {
            cur_index += 1;
        }
        partition[i] = cur_index;
    }
    return partition;
}

int * creat_mat(int dimX, int dimY)
{
    int x;
    int * mat = (int *) malloc(dimX * dimY * sizeof(int));

    srand(time(NULL));

    for (x = 0; x < dimX * dimY; x++) {
        mat[x] = rand() % MOD_BASE;
        //mat[x] = (rand() % MAX_DIM) * SCALING_FACTOR;
        //printf("%d ", mat[x]);
    }
    return mat;
}

void if_mats_equal(int * A,  int * B, int rows, int cols)
{
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (A[i * cols + j] != B[i * cols + j]) { 
                printf("Matrices are not equal\n"); 
                //printf("%d %d\n", i, j);
                return;
            }
        }
    }
    printf("Matrices are equal\n");
}

void cpu_mat_mul(int* A, int* B, int* C, int ARows, int ACols, int BRows, int BCols)
{
    int sum = 0;
    for (int i = 0; i < ARows; i++) {
        for (int j = 0; j < BCols; j++) {
            for (int k = 0; k < ACols; k++) {
                sum += (A[i * ACols + k] * B[k * BCols + j]) % MOD_BASE;
                //sum += A[i * ACols + k] * B[k * BCols + j];
            }
            //printf("%d %d\n", i, j);
            C[i * BCols + j] = sum % MOD_BASE;
            //C[i * BCols + j] = sum;
            sum = 0;
        }
    }
}

void print_mat(int * mat, int dimX, int dimY)
{
    for (int i = 0; i < dimX; i++) {
        for (int j = 0; j < dimY; j++) {
            printf("%d  ", mat[i * dimY + j]);
        }
        printf("\n");
    }
}

int * cpu_multi_mat_mult(int num_dim, int * dim_list, int ** mat_list, int start = 0) {

    int max_dim = find_max(dim_list, num_dim);
    int * output_mat1 = (int *) calloc(max_dim * max_dim, sizeof(int));
    int * output_mat2 = (int *) calloc(max_dim * max_dim, sizeof(int));

    cpu_mat_mul(mat_list[start], mat_list[start + 1], output_mat1, dim_list[start], dim_list[start + 1], dim_list[start + 1], dim_list[start + 2]);
    int num_rows = dim_list[start];
    int num_cols = dim_list[start + 2];

    //print_mat(output_mat1, num_rows, num_cols);

    int num_mult;
    for (num_mult = 1; num_mult < num_dim - 2; num_mult++) {
        //printf("multiplied %d matrices\n", num_mult + 1);
        if (num_mult % 2 == 1) {
            cpu_mat_mul(output_mat1, mat_list[start + num_mult + 1], output_mat2, num_rows, num_cols, dim_list[start + num_mult + 1] , dim_list[start + num_mult + 2]);
        }
        else {
            cpu_mat_mul(output_mat2, mat_list[start + num_mult + 1], output_mat1, num_rows, num_cols, dim_list[start + num_mult + 1] , dim_list[start + num_mult + 2]);
        }
        num_cols = dim_list[start + num_mult + 2];
    }

    //printf("%d %d\n", num_rows, num_cols);
    if (num_mult % 2 == 1) {
        free(output_mat2);
        return output_mat1;
    }
    else {
        free(output_mat1);
        return output_mat2;
    }
}

__device__
void matmult(int* A, int* B, int* C, int ARows, int ACols, int BRows, int BCols)
{
    int num_elem_output = ARows * BCols;
    int C_elem_row = 0;
    int C_elem_col = 0;
    int sum = 0;

    for (int n = threadIdx.x; n < num_elem_output; n+=NUM_THREADS) {
        C_elem_col = n % BCols;
        C_elem_row = (n + (BCols - C_elem_col)) / BCols - 1;
        
        for (int i = 0; i < ACols; i++) {
            sum += (A[C_elem_row * ACols + i] * B[i * BCols + C_elem_col]) % MOD_BASE;
            //sum += A[C_elem_row * ACols + i] * B[i * BCols + C_elem_col];
        }
        C[C_elem_row * BCols + C_elem_col] = sum % MOD_BASE;
        //C[C_elem_row * BCols + C_elem_col] = sum;
        sum = 0;
        __syncthreads();
    }
    __syncthreads();
}

__global__
void gpu_seq_multi_matmult(int num_dim, int * dim_list, int ** mat_list, int * output_mat1, int * output_mat2)
{

    matmult(mat_list[0], mat_list[1], output_mat1, dim_list[0], dim_list[1], dim_list[1], dim_list[2]);
    __syncthreads();

    int num_mult;
    int num_rows = dim_list[0];
    int num_cols = dim_list[2];
    for (num_mult = 1; num_mult < num_dim - 2; num_mult++) {
        if (num_mult % 2 == 1) {
            matmult(output_mat1, mat_list[num_mult + 1], output_mat2, num_rows, num_cols, dim_list[num_mult + 1], dim_list[num_mult + 2]);
        } else {
            matmult(output_mat2, mat_list[num_mult + 1], output_mat1, num_rows, num_cols, dim_list[num_mult + 1], dim_list[num_mult + 2]);
        }
        num_cols = dim_list[num_mult + 2];
        __syncthreads();
    }
}

__global__
void gpu_par_multi_matmult(int start_dim_idx, int end_dim_idx, int * dim_list, int ** mat_list, int * output_mat1, int * output_mat2) //, int* i)
{

    matmult(mat_list[start_dim_idx], mat_list[start_dim_idx + 1], output_mat1, dim_list[start_dim_idx], dim_list[start_dim_idx + 1], dim_list[start_dim_idx + 1], dim_list[start_dim_idx + 2]);
/*
    if (threadIdx.x == 0) { 
        i[0]++; 
        i[1] = start_dim_idx;
        i[2] = end_dim_idx;
        }
*/
    __syncthreads();


    int num_mult;
    int num_rows = dim_list[start_dim_idx];
    int num_cols = dim_list[start_dim_idx + 2];
    int count = 1;
//    for (num_mult = start_dim_idx + 1; count < end_dim_idx - start_dim_idx - 2; num_mult++) {
    for (int count = 1; count < (end_dim_idx - start_dim_idx - 2); count++) {
        if (count % 2 == 1) {
            matmult(output_mat1, mat_list[start_dim_idx + count + 1], output_mat2, num_rows, num_cols, dim_list[start_dim_idx + count + 1], dim_list[start_dim_idx + count + 2]);
        } else {
            matmult(output_mat2, mat_list[start_dim_idx + count + 1], output_mat1, num_rows, num_cols, dim_list[start_dim_idx + count + 1], dim_list[start_dim_idx + count + 2]);
            //matmult(output_mat2, mat_list[num_mult + 1], output_mat1, num_rows, num_cols, dim_list[num_mult + 1], dim_list[num_mult + 2]);
        }
        num_cols = dim_list[start_dim_idx + count + 2];
/*        
        if (threadIdx.x == 0) { 
            i[0]++; 
            i[1] = num_rows;
            i[2] = num_cols;
            }
*/
        __syncthreads();
    }

}


int main()
{

    int nblocks = 1;
    int num_dim = 45;
    int num_mat = num_dim - 1;
    int * mat_dim = def_mat_dim(num_dim);
    int ** mat_list = (int **) malloc((num_mat) * sizeof(int *));
    int max_dim = find_max(mat_dim, num_dim);

    printf("Copying matrix dimensions to device\n");

    int * d_mat_dim;
    hipMalloc((void **)&d_mat_dim, num_dim * sizeof(int));
    hipMemcpy(d_mat_dim, mat_dim, num_dim * sizeof(int), hipMemcpyHostToDevice);

    printf("Creating Matrix from on host\n");

    int k;
    for (k = 0; k < num_mat; k++) {
        //printf("================= MATRIX %d ====================\n", k);
        //printf("%d %d\n", mat_dim[k], mat_dim[k+1]);
        mat_list[k] = creat_mat(mat_dim[k], mat_dim[k+1]);
        //printf("%d %d\n", mat_dim[k], mat_dim[k+1]);
        //print_mat(mat_list[k], mat_dim[k], mat_dim[k+1]);
    }
    
    printf("Allocating space to store output matrix\n");
    int * out_mat = (int *) calloc(max_dim * max_dim, sizeof(int));
    int * d_out_mat1, * d_out_mat2;
    hipMalloc((void **) &d_out_mat1, max_dim * max_dim * sizeof(int));
    hipMalloc((void **) &d_out_mat2, max_dim * max_dim * sizeof(int));
    hipMemcpy(d_out_mat1, out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_mat2, out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);

    printf("Allocating space for each matrix, and storing pointer address of matrices on the host\n");
    int ** int_mat_list = (int **) malloc(num_mat * sizeof(int *));
    for (k = 0; k < num_mat; k++) {
        hipMalloc((void **)&int_mat_list[k], mat_dim[k] * mat_dim[k+1] * sizeof(int));
        hipMemcpy(int_mat_list[k], mat_list[k], mat_dim[k] * mat_dim[k+1] * sizeof(int), hipMemcpyHostToDevice);
    }

    printf("Copying pointer addresses of matrices from host to device\n");
    int ** d_mat_list;
    hipMalloc(&d_mat_list, num_mat * sizeof(int *));
    hipMemcpy(d_mat_list, int_mat_list, num_mat * sizeof(int *), hipMemcpyHostToDevice);


    printf("Allocating a set of intermediate arrays\n");
//  Allocating a set of intermediate 
    int ** int_mat1, ** int_mat2;
    int_mat1 = (int **) malloc(nblocks * sizeof(int *));
    int_mat2 = (int **) malloc(nblocks * sizeof(int *));
    for (k = 0; k < nblocks; k++) {
        hipMalloc((void **)&int_mat1[k], max_dim * max_dim * sizeof(int));
        hipMalloc((void **)&int_mat2[k], max_dim * max_dim * sizeof(int));
        hipMemcpy(int_mat1[k], out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(int_mat2[k], out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);
    }

    printf("allocating final collection of intermediate arrays\n");
    int ** int_mat_final = (int **) malloc(nblocks * sizeof(int *));
    int ** d_int_mat_final;
    hipMalloc((void **)&d_int_mat_final, nblocks * sizeof(int *));


    int * mat_list_partition = equipartition(num_dim, nblocks);
    printf("partition\n");
    print_mat(mat_list_partition, 1, nblocks + 1);

    printf("mat_dim\n");
    print_mat(mat_dim, 1, num_dim);
    int * int_mat_dim = (int *) malloc((nblocks + 1) * sizeof(int));
    for (int i = 0; i < nblocks + 1; i++) {
        int_mat_dim[i] = mat_dim[mat_list_partition[i]];
    }
    printf("partition mat_dim\n");
    print_mat(int_mat_dim, 1, nblocks + 1);
    printf("\n");
    int * d_int_mat_dim;
    hipMalloc((void **)&d_int_mat_dim, (nblocks + 1) * sizeof(int));
    hipMemcpy(d_int_mat_dim, int_mat_dim, (nblocks + 1) * sizeof(int), hipMemcpyHostToDevice);

    int * d_int_output_mat1, * d_int_output_mat2;
    hipMalloc((void **)&d_int_output_mat1, max_dim * max_dim * sizeof(int));
    hipMalloc((void **)&d_int_output_mat2, max_dim * max_dim * sizeof(int));
    hipMemcpy(d_int_output_mat1, out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_int_output_mat2, out_mat, max_dim * max_dim * sizeof(int), hipMemcpyHostToDevice);



    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
       // print the CUDA error message and exit
       printf("CUDA error: %s\n", hipGetErrorString(error));
       exit(-1);
    }

    int * cpu_mat = cpu_multi_mat_mult(num_dim, mat_dim, mat_list);
    //printf("%d %d\n", mat_dim[0], mat_dim[num_dim-1]);
    printf("%d %d\n", mat_dim[0], mat_dim[num_dim-1]);
    printf("cpu seq\n");
    print_mat(cpu_mat, mat_dim[0], mat_dim[num_dim-1]);
    printf("\n");

    //printf("%d %d %d\n", mat_dim[0], mat_dim[1], mat_dim[2]);
    //matmult<<<dimGrid, dimBlock>>>(int_mat_list[0], int_mat_list[1], d_out_mat, mat_dim[0], mat_dim[1], mat_dim[1], mat_dim[2]);
    //matmult<<<1, NUM_THREADS>>>(int_mat_list[0], int_mat_list[1], d_out_mat, mat_dim[0], mat_dim[1], mat_dim[1], mat_dim[2]);
    //cudaThreadSynchronize();
    //multi_matmult<<<1, NUM_THREADS>>>(num_dim, d_mat_dim, d_mat_list, d_out_mat);
    //gpuErrchk(cudaPeekAtLastError());

    //gpu_seq_multi_matmult<<<1, NUM_THREADS>>>(num_dim, d_mat_dim, d_mat_list, d_out_mat1, d_out_mat2);
    //cudaThreadSynchronize();

/*
    int ** tmp = (int **) malloc((num_dim - 1) * sizeof(int *));
    cudaMemcpy(tmp, d_mat_list, num_mat * sizeof(int *), cudaMemcpyDeviceToHost);
    for (int i = 0; i < num_dim - 1; i++) {
        printf("%p\n", tmp[i]);
        cudaMemcpy(out_mat, tmp[i], mat_dim[i] * mat_dim[i+1] * sizeof(int), cudaMemcpyDeviceToHost);
        print_mat(out_mat, mat_dim[i], mat_dim[i+1]);

    }
*/
/*
    int iii[] = {0, 0, 0, 0, 0};
    int * d_iii;
    cudaMalloc((void **)&d_iii,5 *  sizeof(int));
    cudaMemcpy(d_iii, iii, 5 * sizeof(int), cudaMemcpyHostToDevice);
*/
    for (int i = 0; i < nblocks; i++) {

        printf("nblocks %d, i %d\n", nblocks, i);
        printf("mat_list_partition %d %d\n", mat_list_partition[i], mat_list_partition[i+1] - 1);
        printf("end mat_dim %d %d\n", mat_dim[mat_list_partition[i]], mat_dim[mat_list_partition[i+1]]);
        gpu_par_multi_matmult<<<1, NUM_THREADS>>>(mat_list_partition[i], mat_list_partition[i+1] + 1, d_mat_dim, d_mat_list, int_mat1[i], int_mat2[i]); //, d_iii);
        //cudaMemcpy(iii, d_iii, 5 * sizeof(int), cudaMemcpyDeviceToHost);
        //printf("num_products %d\n", iii[0]);
        //printf("num_rows %d\n", iii[1]);
        //printf("num_cols %d\n", iii[2]);
    }
    hipDeviceSynchronize();

// Break up case for when only one block is chosen, and when many blocks (more than 1) is chosen

    //print_mat((int*)int_mat1, 1, nblocks);
    //print_mat((int*)int_mat2, 1, nblocks);
    for (int i = 0; i < nblocks; i++) {
        hipMemcpy(out_mat, int_mat1[i], mat_dim[mat_list_partition[i]] * mat_dim[mat_list_partition[i+1]] * sizeof(int), hipMemcpyDeviceToHost);
        printf("printing int_mat1[%d]\n", i);
        print_mat(out_mat, mat_dim[mat_list_partition[i]], mat_dim[mat_list_partition[i+1]]);
        hipMemcpy(out_mat, int_mat2[i], mat_dim[mat_list_partition[i]] * mat_dim[mat_list_partition[i+1]] * sizeof(int), hipMemcpyDeviceToHost);
        printf("printing int_mat2[%d]\n", i);
        print_mat(out_mat, mat_dim[mat_list_partition[i]], mat_dim[mat_list_partition[i+1]]);
        if ((mat_list_partition[i+1] - mat_list_partition[i]) % 2 == 1) {
            int_mat_final[i] = int_mat1[i];
        } else {
            int_mat_final[i] = int_mat2[i];
        }
    }
    //print_mat((int*)int_mat_final, 1, nblocks);
    //cudaMemcpy(out_mat, int_mat_final, mat_dim[mat_list_partition[i]] * mat_dim[mat_list_partition[i]] * sizeof(int), cudaMemcpyDeviceToHost);

    hipMemcpy(d_int_mat_final, int_mat_final, nblocks * sizeof(int *), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    
    printf("Calling last kernel\n");
    gpu_par_multi_matmult<<<1, NUM_THREADS>>>(0, nblocks + 1, d_int_mat_dim, d_int_mat_final, d_int_output_mat1, d_int_output_mat2); //, iii);
    hipDeviceSynchronize();


    error = hipGetLastError();
    if(error != hipSuccess)
    {
       // print the CUDA error message and exit
       printf("CUDA error: %s\n", hipGetErrorString(error));
       exit(-1);
    }

    if (nblocks % 2 == 1) {
        hipMemcpy(out_mat, d_int_output_mat1, mat_dim[0] * mat_dim[num_dim-1] * sizeof(int), hipMemcpyDeviceToHost);   
    } else {
        hipMemcpy(out_mat, d_int_output_mat2, mat_dim[0] * mat_dim[num_dim-1] * sizeof(int), hipMemcpyDeviceToHost);   
    }

    printf("%d %d\n", mat_dim[mat_list_partition[0]], mat_dim[mat_list_partition[nblocks]]);
    printf("gpu par\n");
    print_mat(out_mat, mat_dim[0], mat_dim[num_dim-1]);
    printf("\n");

    if_mats_equal(out_mat, cpu_mat, mat_dim[0], mat_dim[num_dim-1]);
/*    
    if (num_dim % 2 == 1) {
        cudaMemcpy(out_mat, d_out_mat1, mat_dim[0] * mat_dim[num_dim-1] * sizeof(int), cudaMemcpyDeviceToHost);
    } else {
        cudaMemcpy(out_mat, d_out_mat2, mat_dim[0] * mat_dim[num_dim-1] * sizeof(int), cudaMemcpyDeviceToHost);
    }

    printf("gpu seq\n");
    print_mat(out_mat, mat_dim[0], mat_dim[num_dim-1]);
    printf("\n");

    if_mats_equal(out_mat, cpu_mat, mat_dim[0], mat_dim[num_dim-1]);
*/    
    return 0;

}
